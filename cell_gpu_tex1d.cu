#include "hip/hip_runtime.h"
#include <stdio.h>
#include "para.h"
#include <ctime>

texture<int> texIn;
texture<int> texOut;

// the kernel to udpate the state of each cell based on its neighbors for certain number of steps
__global__ void update_tex1d(int *out, int flag,int dim){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	while (offset < dim * dim) {
	int sum = 0;
	int origin;
	
	for(int i = -1; i < 2; i++){
		for(int j = -1; j < 2; j++){
			int xtemp = (x + i + dim) % dim;
			int ytemp = (y + j + dim) % dim;
			int offsettemp=xtemp + ytemp*blockDim.x*gridDim.x;
			if (flag == 1)			
				sum = sum + tex1Dfetch(texIn, offsettemp);
			else
				sum = sum + tex1Dfetch(texOut, offsettemp);
		}
	}
	if (flag == 1)			
		origin=tex1Dfetch(texIn,offset);
	else
		origin=tex1Dfetch(texOut,offset);
	
	sum=sum-origin;
	if (origin == 1) {
		if(sum == 2 || sum ==3) out[offset] = 1;
		else out[offset] = 0;
	}
	else{
		if(sum == 3) out[offset] = 1;
		else out[offset] = 0;
	}
 	offset = offset + blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    }
}
	
int main(int argc, char *argv[]){
    clock_t start;
    clock_t gpu_start;
 float gpu_comp_time = 0;
    float gpu_mem_to_time = 0, gpu_mem_back_time=0;

	int dim = atoi(argv[1]);
	int nStep = atoi(argv[2]);
	int frequency = atoi(argv[3]);
	int size = dim * dim;
	int step;
	DataBlock data;
data.bitmap=(int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        data.bitmap[i] = 0;
    }
    data.bitmap[1] = 1;
    data.bitmap[dim + 2] = 1;
    data.bitmap[2 * dim + 0] = 1;
    data.bitmap[2 * dim + 1] = 1;
    data.bitmap[2 * dim + 2] = 1;
    data.outbitmap=(int *)malloc(size * sizeof(int));
    int bitmapSize=size * sizeof(int);
    
  start=clock();

    gpu_start = clock();


	int flag;
	HANDLE_ERROR(hipMalloc( (void **)&(data.dev_in), bitmapSize));
	HANDLE_ERROR(hipMalloc( (void **)&(data.dev_out), bitmapSize));

	
	HANDLE_ERROR(hipBindTexture (NULL, texIn, data.dev_in, bitmapSize));
	HANDLE_ERROR(hipBindTexture (NULL, texOut, data.dev_out, bitmapSize));



	HANDLE_ERROR(hipMemcpy(data.dev_in, data.bitmap, bitmapSize, hipMemcpyHostToDevice));

 gpu_mem_to_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;

	dim3 dimgrid(dim/16,dim/16);
	dim3 dimblock(16,16);

 gpu_start = clock();
	flag = 1;
	for(step = 0; step<nStep; step++ ){
		int *in,*out;

		if( flag == 1 ) {
			out = data.dev_out;flag = 0;
		}
		else {
			out=data.dev_in; flag = 1;
		}
// gpu_start = clock();
		update_tex1d<<<dimgrid,dimblock>>>(out, flag, dim);
//	  gpu_comp_time += ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
//		swap(data.dev_in,data.dev_out);

//		if(step % frequency == frequency - 1){
//  gpu_start = clock();
//			HANDLE_ERROR(hipMemcpy(data.outbitmap,data.dev_out,bitmapSize,hipMemcpyDeviceToHost));
 // gpu_mem_time += ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
//			printf ("\nIteration %d / %d : \n", step + 1, nStep);
//			for (int j = 0; j < size; j++) {
//				if ( j % dim == 0 ) {
//					printf("\n");
//				}
//				printf ("%d  ", data.outbitmap[j]);
//			}
//			printf("\n");
		}
    }

	  gpu_comp_time += ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
  gpu_start = clock();

	HANDLE_ERROR(hipMemcpy(data.outbitmap,data.dev_out,bitmapSize,hipMemcpyDeviceToHost));
  gpu_mem_back_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
	HANDLE_ERROR(hipFree(data.dev_in));
	HANDLE_ERROR(hipFree(data.dev_out));
    printf("%f %f %f", gpu_comp_time, gpu_mem_to_time, gpu_mem_back_time);
    printf("%f\n", ((float)(clock() - start)) / CLOCKS_PER_SEC);
}
