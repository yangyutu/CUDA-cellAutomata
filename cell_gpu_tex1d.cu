#include "hip/hip_runtime.h"
#include <stdio.h>
#include "para.h"

texture<int> texIn;
texture<int> texOut;

// the kernel to udpate the state of each cell based on its neighbors for certain number of steps
__global__ void update_tex1d(int *out, int flag,int dim){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y*blockDim.x*gridDim.x;
	int sum=0;
	int origin;
	
	for(int i=-1;i<2;i++){
		for(int j=-1;j<2;j++){
			int xtemp=(x+i+dim)%dim;
			int ytemp=(y+j+dim)%dim;
			int offsettemp=xtemp + ytemp*blockDim.x*gridDim.x;
			if (flag == 1)			
				sum=sum+tex1Dfetch(texIn,offsettemp);
			else
				sum=sum+tex1Dfetch(texOut,offsettemp);
		}
	}
			if (flag == 1)			
				origin=tex1Dfetch(texIn,offset);
			else
				origin=tex1Dfetch(texOut,offset);
	
	sum=sum-origin;
	if (origin == 1) {
		if(sum == 2 || sum ==3) out[offset] = 1;
		else out[offset] = 0;
	}else{
	if( sum == 3) out[offset] = 1;
	}

}
	
int main(int argc, char *argv[]){

	int dim=100;
	int nStep=1000;
	int frequency=100;
	int size=dim*dim;
	int step;
	DataBlock data;
	data.bitmap=(int *)malloc(size*sizeof(int));
	int bitmapSize=size*sizeof(int);
	int flag;
	HANDLE_ERROR(hipMalloc( (void **)&(data.dev_in),bitmapSize));
	HANDLE_ERROR(hipMalloc( (void **)&(data.dev_out),bitmapSize));
//	HANDLE_ERROR(hipMalloc( (void **)&(data.dev_in),size));
	
	HANDLE_ERROR(hipBindTexture (NULL, texIn, data.dev_in, bitmapSize));
	HANDLE_ERROR(hipBindTexture (NULL, texOut, data.dev_out, bitmapSize));



	HANDLE_ERROR(hipMemcpy(data.dev_in, data.bitmap, bitmapSize, hipMemcpyHostToDevice));

	dim3 dimgrid(dim/16,dim/16);
	dim3 dimblock(16,16);

	flag = 1;
	for(step = 0; step<nStep; step++ ){
	int *in,*out;

	if( flag == 1 ) {out = data.dev_out;flag = 0;}
	else {out=data.dev_in; flag = 1;}

	
	update_tex1d<<<dimgrid,dimblock>>>(out,flag,dim);
	
//	swap(data.dev_in,data.dev_out);

	if(step % frequency == 0){
	HANDLE_ERROR(hipMemcpy(data.outbitmap,data.dev_out,bitmapSize,hipMemcpyDeviceToHost));

      printf ( "\nIteration %d: final grid:\n", step );
      for ( int j = 0; j < size; j++ ) {
        if ( j % dim == 0 ) {printf( "\n" );}
        printf ( "%d  ", data.outbitmap[j] );
      }
      printf( "\n" );
	}
    	}

	HANDLE_ERROR(hipFree(data.dev_in));
	HANDLE_ERROR(hipFree(data.dev_out));

}
