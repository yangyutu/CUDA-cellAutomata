#include "hip/hip_runtime.h"
#include <stdio.h>
#include "para.h"
#include <ctime>


// the kernel to udpate the state of each cell based on its neighbors for certain number of steps
__global__ void update(int *in, int *out, int dim){
    
    int offset = threadIdx.x + blockIdx.x * blockDim.x;
    int x = offset % dim;
    int y = (int)(offset / dim);
    while (offset < dim * dim) {
        int sum = 0;
        for(int i=-1; i < 2; i++) {
            for(int j=-1; j < 2; j++) {
                int xtemp = (x + i + dim) % dim;
                int ytemp = (y + j + dim) % dim;
                int offsettemp = xtemp + ytemp * blockDim.x;
                sum = sum + in[offsettemp];
            }
        }
        sum = sum - in[offset];
        if (in[offset] == 1) {
            if (sum == 2 || sum ==3) { 
                out[offset] = 1;
            }
            else { 
                out[offset] = 0;
            }
        }
        else {
            if( sum == 3) {
                out[offset] = 1;
            }
            else {
                out[offset] = 0;
            }
        }
        offset = offset + blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    }
}

int main(int argc, char *argv[]) {

    clock_t start;
    clock_t gpu_start;
    float gpu_comp_time = 0;
    float gpu_mem_to_time = 0, gpu_mem_back_time=0;
    int dim = atoi(argv[1]);
    int nStep = atoi(argv[2]);
    int frequency = atoi(argv[3]);
    int size = dim * dim;
    int step;
    DataBlock data;
    data.bitmap=(int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        data.bitmap[i] = 0;
    }
    data.bitmap[1]=1;
    data.bitmap[dim+2] = 1;
    data.bitmap[2 * dim + 0] = 1;
    data.bitmap[2 * dim + 1] = 1;
    data.bitmap[2 * dim + 2] = 1;
    data.outbitmap=(int *)malloc(size * sizeof(int));
    int bitmapSize=size * sizeof(int);
    
    start=clock();

    gpu_start = clock();
    HANDLE_ERROR(hipMalloc( (void **)&(data.dev_in), bitmapSize));
    HANDLE_ERROR(hipMalloc( (void **)&(data.dev_out), bitmapSize));
    HANDLE_ERROR(hipMemcpy(data.dev_in, data.bitmap, bitmapSize, hipMemcpyHostToDevice));
 gpu_mem_to_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;

    // dim3 dimgrid(dim / 16, dim / 16);
    // dim3 dimblock(16, 16);
         gpu_start = clock();   
    for(step = 0; step < nStep; step++ ){


        update<<<dim,dim>>>(data.dev_in, data.dev_out,dim);
  

        swap(data.dev_in,data.dev_out);
 //       if(step % frequency == frequency - 1 ){
 //           gpu_start = clock();
 //           HANDLE_ERROR(hipMemcpy(data.outbitmap, data.dev_out, bitmapSize, hipMemcpyDeviceToHost));
 //           gpu_mem_time += ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
            // printf ( "\nIteration %d: final grid:\n", step );
            // for (int j = 0; j < size; j++) {
            //     if ( j % dim == 0 ) {
            //         printf( "\n" );
            //     }
            //     printf("%d", data.outbitmap[j]);
            // }
            // printf( "\n" );
//        }
    }
	hipDeviceSynchronize();
      gpu_comp_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
            gpu_start = clock();
            HANDLE_ERROR(hipMemcpy(data.outbitmap, data.dev_out, bitmapSize, hipMemcpyDeviceToHost));
            gpu_mem_back_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
    HANDLE_ERROR(hipFree(data.dev_out));
    HANDLE_ERROR(hipFree(data.dev_in));
printf("%f %f %f ", gpu_comp_time, gpu_mem_to_time, gpu_mem_back_time);
    printf("%f\n", ((float)(clock() - start)) / CLOCKS_PER_SEC);
}
