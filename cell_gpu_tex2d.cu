#include "hip/hip_runtime.h"

#include <stdio.h>
#include "para.h"
#include <ctime>
#include "hip/hip_runtime.h"

texture<int,2> texIn;
texture<int,2> texOut;

// the kernel to udpate the state of each cell based on its neighbors for certain number of steps
__global__ void update_tex1d(int *out, int flag,int dim){
    
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    
    while (offset < dim * dim) {
        int sum = 0;
        int origin;
        
        for(int i = -1; i < 2; i++){
            for(int j = -1;j < 2; j++){
                int xtemp = (x + i + dim) % dim;
                int ytemp = (y + j + dim) % dim;
                int offsettemp = xtemp + ytemp * blockDim.x * gridDim.x;
                if (flag == 1)
                    sum=sum+tex2D(texIn,xtemp,ytemp);
                else
                    sum=sum+tex2D(texOut,xtemp,ytemp);
            }
        }
        if (flag == 1)
            origin = tex2D(texIn, x, y);
        else
            origin = tex2D(texOut, x, y);
        
        sum = sum - origin;
        if (origin == 1) {
            if(sum == 2 || sum ==3) out[offset] = 1;
            else out[offset] = 0;
        }
        else{
            if( sum == 3) out[offset] = 1;
            else out[offset] = 0;
        }
        offset = offset + blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    }
}

int main(int argc, char *argv[]){
    clock_t start;
    clock_t gpu_start;
    float gpu_comp_time = 0;
    float gpu_mem_to_time = 0, gpu_mem_back_time=0;
    
    int dim = atoi(argv[1]);
    int nStep = atoi(argv[2]);
//    int frequency = atoi(argv[3]);
    int size = dim * dim;
    int step;
    DataBlock data;
    data.bitmap = (int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        data.bitmap[i] = 0;
    }
    data.bitmap[1] = 1;
    data.bitmap[dim + 2] = 1;
    data.bitmap[2 * dim + 0] = 1;
    data.bitmap[2 * dim + 1] = 1;
    data.bitmap[2 * dim + 2] = 1;
    data.outbitmap = (int *)malloc(size * sizeof(int));
    int bitmapSize = size * sizeof(int);
    
    
    start=clock();
    
    gpu_start = clock();
    
    int flag;
    HANDLE_ERROR(hipMalloc((void **)&(data.dev_in), bitmapSize));
    HANDLE_ERROR(hipMalloc((void **)&(data.dev_out), bitmapSize));
    
    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    HANDLE_ERROR(hipBindTexture2D(NULL, texIn, data.dev_in, desc, dim, dim, sizeof(int) * dim));
    HANDLE_ERROR(hipBindTexture2D(NULL, texOut, data.dev_out, desc, dim, dim, sizeof(int) * dim));
    
    
    
    HANDLE_ERROR(hipMemcpy(data.dev_in, data.bitmap, bitmapSize, hipMemcpyHostToDevice));
    
    gpu_mem_to_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
    
    dim3 dimgrid(dim / 16, dim / 16);
    dim3 dimblock(16, 16);
    gpu_start = clock();
    flag = 1;
    
    for(step = 0; step < nStep; step++ ){
        int *in,*out;
        
        if( flag == 1 ) {
            out = data.dev_out;flag = 0;
        }
        else {
            out=data.dev_in; flag = 1;
        }
        
        update_tex1d<<<dimgrid,dimblock>>>(out,flag,dim);

    }
    hipDeviceSynchronize();
    gpu_comp_time += ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
    gpu_start = clock();
    gpu_mem_back_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
    HANDLE_ERROR(hipFree(data.dev_in));
    HANDLE_ERROR(hipFree(data.dev_out));
    
    printf("%f %f %f ", gpu_comp_time, gpu_mem_to_time, gpu_mem_back_time);
    printf("%f\n", ((float)(clock() - start)) / CLOCKS_PER_SEC);
}
